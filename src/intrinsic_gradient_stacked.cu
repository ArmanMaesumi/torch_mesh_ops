#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <math.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAException.h>

// Each thread computes the gradient block for one face.
// Stacked variant: rows are [0..F-1] for x-components, [F..2F-1] for y-components.
template <typename scalar_t, typename index_t>
__global__ void intrinsic_gradient_kernel_stacked(
    const scalar_t* __restrict__ edge_lengths, // shape: (F, 3)
    const index_t*  __restrict__ faces,        // shape: (F, 3)
    int64_t* __restrict__ indices,             // shape: (2, 6F)
    scalar_t* __restrict__ values,             // shape: (6F)
    const int64_t F
) {
  const int64_t f = blockIdx.x * blockDim.x + threadIdx.x;
  if (f >= F) return;

  // Read the three edge lengths for face f.
  const scalar_t l0 = edge_lengths[f * 3 + 0];
  const scalar_t l1 = edge_lengths[f * 3 + 1];
  const scalar_t l2 = edge_lengths[f * 3 + 2];

  // Build a 2D triangle in intrinsic space
  const scalar_t x = (l1 * l1 + l2 * l2 - l0 * l0) / (static_cast<scalar_t>(2) * l2);
  const scalar_t tmp = l1 * l1 - x * x;
  const scalar_t y = sqrt(tmp > 0 ? tmp : 0);
  const scalar_t A = static_cast<scalar_t>(0.5) * l2 * y;  // triangle area
  const scalar_t inv2A = static_cast<scalar_t>(1.0) / (static_cast<scalar_t>(2.0) * A);

  // Barycentric gradients
  const scalar_t g0_x = -y * inv2A;
  const scalar_t g0_y = (x - l2) * inv2A;
  const scalar_t g1_x =  y * inv2A;
  const scalar_t g1_y = -x * inv2A;
  const scalar_t g2_x = static_cast<scalar_t>(0);
  const scalar_t g2_y = l2 * inv2A;

  // Face vertex indices
  const int64_t v0 = static_cast<int64_t>(faces[f * 3 + 0]);
  const int64_t v1 = static_cast<int64_t>(faces[f * 3 + 1]);
  const int64_t v2 = static_cast<int64_t>(faces[f * 3 + 2]);

  // Each face contributes 6 entries (2 rows × 3 vertices).
  const int64_t base = f * 6;
  const int64_t rows_stride = F * 6;

  // Stacked row indices
  const int64_t row_x = f;      // x rows first
  const int64_t row_y = F + f;  // y rows stacked after all x

  // Fill row indices
  indices[0 * rows_stride + base + 0] = row_x; // g0_x (v0)
  indices[0 * rows_stride + base + 1] = row_x; // g1_x (v1)
  indices[0 * rows_stride + base + 2] = row_x; // g2_x (v2)
  indices[0 * rows_stride + base + 3] = row_y; // g0_y (v0)
  indices[0 * rows_stride + base + 4] = row_y; // g1_y (v1)
  indices[0 * rows_stride + base + 5] = row_y; // g2_y (v2)

  // Fill column indices
  indices[1 * rows_stride + base + 0] = v0;
  indices[1 * rows_stride + base + 1] = v1;
  indices[1 * rows_stride + base + 2] = v2;
  indices[1 * rows_stride + base + 3] = v0;
  indices[1 * rows_stride + base + 4] = v1;
  indices[1 * rows_stride + base + 5] = v2;

  // Values
  values[base + 0] = g0_x;
  values[base + 1] = g1_x;
  values[base + 2] = g2_x;
  values[base + 3] = g0_y;
  values[base + 4] = g1_y;
  values[base + 5] = g2_y;
}

static inline void intrinsic_gradient_cuda_stacked(
    torch::Tensor edge_lengths,  // (F, 3)
    torch::Tensor faces,         // (F, 3)
    torch::Tensor indices,       // (2, 6F)
    torch::Tensor values         // (6F)
) {
  const int64_t F = edge_lengths.size(0);
  const int threads = 256;
  const int blocks = static_cast<int>((F + threads - 1) / threads);

  c10::cuda::CUDAGuard guard(edge_lengths.device());
  auto stream = at::cuda::getCurrentCUDAStream(edge_lengths.device().index());

  AT_DISPATCH_FLOATING_TYPES(edge_lengths.scalar_type(), "intrinsic_gradient_cuda_stacked", [&] {
    using scalar_t_ = scalar_t;
    if (faces.scalar_type() == at::kLong) {
      intrinsic_gradient_kernel_stacked<scalar_t_, int64_t>
        <<<blocks, threads, 0, stream.stream()>>>(
          edge_lengths.data_ptr<scalar_t_>(),
          faces.data_ptr<int64_t>(),
          indices.data_ptr<int64_t>(),
          values.data_ptr<scalar_t_>(),
          F);
    } else if (faces.scalar_type() == at::kInt) {
      intrinsic_gradient_kernel_stacked<scalar_t_, int32_t>
        <<<blocks, threads, 0, stream.stream()>>>(
          edge_lengths.data_ptr<scalar_t_>(),
          faces.data_ptr<int32_t>(),
          indices.data_ptr<int64_t>(),
          values.data_ptr<scalar_t_>(),
          F);
    } else {
      TORCH_CHECK(false, "faces must be int32 or int64");
    }
  });
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

// This function computes the intrinsic gradient operator (stacked rows) and returns a sparse tensor.
torch::Tensor intrinsic_gradient_operator_stacked(
  torch::Tensor edge_lengths, // (F, 3)
  torch::Tensor faces         // (F, 3) int32/int64
) {
  TORCH_CHECK(edge_lengths.is_cuda(), "edge_lengths must be a CUDA tensor");
  TORCH_CHECK(faces.is_cuda(), "faces must be a CUDA tensor");
  TORCH_CHECK(edge_lengths.dim() == 2 && edge_lengths.size(1) == 3,
              "edge_lengths must be of shape (F, 3)");
  TORCH_CHECK(faces.dim() == 2 && faces.size(1) == 3,
              "faces must be of shape (F, 3)");

  edge_lengths = edge_lengths.contiguous();
  faces = faces.contiguous();

  const int64_t F = edge_lengths.size(0);
  // Determine number of vertices (V) from the face tensor.
  const int64_t V = faces.max().item<int64_t>() + 1;

  auto options_int = torch::TensorOptions().dtype(torch::kInt64)
                                          .device(edge_lengths.device());
  torch::Tensor indices = torch::empty({2, F * 6}, options_int);

  auto options_float = torch::TensorOptions().dtype(edge_lengths.dtype())
                                            .device(edge_lengths.device());
  torch::Tensor values = torch::empty({F * 6}, options_float);

  intrinsic_gradient_cuda_stacked(edge_lengths, faces, indices, values);

  // Construct and return the sparse tensor with shape (2F, V)
  std::vector<int64_t> size = {F * 2, V};
  auto grad_operator = torch::sparse_coo_tensor(indices, values, size);
  return grad_operator;
}

void init_intrinsic_gradient_stacked(py::module &m) {
  m.def("intrinsic_gradient_stacked",
        &intrinsic_gradient_operator_stacked,
        "Intrinsic gradient operator (stacked rows, CUDA)");
}

